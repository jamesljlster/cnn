#include "hip/hip_runtime.h"
#include "cnn_cudef.h"

#ifdef __cplusplus
extern "C"
{
#endif

    __global__ void cnn_drop_kernel(float* dst, float* src, int* mask, int size,
                                    float scale)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= size)
        {
            return;
        }

        if (mask[i] > 0)
        {
            dst[i] = src[i] * scale;
        }
        else
        {
            dst[i] = 0;
        }
    }

    void cnn_drop_gpu(float* dst, float* src, int* mask, int size, float scale)
    {
        int blocks = size / CNN_THREAD_PER_BLOCK;
        if (size % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_drop_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, mask, size,
                                                          scale);
    }

    __global__ void cnn_drop_grad_kernel(float* gradDst, float* gradSrc,
                                         int* mask, int size, float scale)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= size)
        {
            return;
        }

        if (mask[i] > 0)
        {
            gradDst[i] = gradSrc[i] * scale;
        }
    }

    void cnn_drop_grad_gpu(float* gradDst, float* gradSrc, int* mask, int size,
                           float scale)
    {
        int blocks = size / CNN_THREAD_PER_BLOCK;
        if (size % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_drop_grad_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(
            gradDst, gradSrc, mask, size, scale);
    }

    __global__ void cnn_map_kernel(float* dst, float* src, int* map, int len)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        int tmpIndex = map[index];
        if (tmpIndex >= 0)
        {
            dst[index] = src[tmpIndex];
        }
    }

    void cnn_map_gpu(float* dst, float* src, int* map, int len)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_map_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, map, len);
    }

    __global__ void cnn_map_inv_kernel(float* dst, float* src, int* map,
                                       int len)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        int tmpIndex = map[index];
        if (tmpIndex >= 0)
        {
            //dst[tmpIndex] += src[index];
            atomicAdd(dst + tmpIndex, src[index]);
        }
    }

    void cnn_map_inv_gpu(float* dst, float* src, int* map, int len)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_map_inv_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, map,
                                                             len);
    }


#ifdef __cplusplus
}
#endif
