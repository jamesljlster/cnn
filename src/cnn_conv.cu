#include "hip/hip_runtime.h"
#include "cnn_cudef.h"

#ifdef __cplusplus
extern "C"
{
#endif

    __global__ void cnn_map_kernel(float* dst, float* src, int* map, int len)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        int tmpIndex = map[index];
        if (tmpIndex >= 0)
        {
            dst[index] = src[tmpIndex];
        }
    }

    void cnn_map_gpu(float* dst, float* src, int* map, int len)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_map_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, map, len);
    }

    __global__ void cnn_map_inv_kernel(float* dst, float* src, int* map,
                                       int len)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        int tmpIndex = map[index];
        if (tmpIndex >= 0)
        {
            atomicAdd(dst + tmpIndex, src[index]);
        }
    }

    void cnn_map_inv_gpu(float* dst, float* src, int* map, int len)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_map_inv_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, map,
                                                             len);
    }

#ifdef __cplusplus
}
#endif
