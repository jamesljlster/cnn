#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "cnn_builtin_math_cu.h"
#include "cnn_cudef.h"

__device__ float max_cu(float src1, float src2)
{
    return (src1 > src2) ? src1 : src2;
}

__global__ void cnn_max_kernel(float* vec, int len, int slice, int stride,
                               int shift)
{
    int sliceIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (sliceIndex >= slice)
    {
        return;
    }

    int head = sliceIndex * stride;
    int cmp = head + shift;
    if (cmp < len)
    {
        vec[head] = max_cu(vec[head], vec[cmp]);
    }
}

void cnn_max_gpu(float* maxPtr, float* vec, int len, float* buf)
{
    // Copy memory
    hipMemcpy(buf, vec, len * sizeof(float), hipMemcpyDeviceToDevice);

    // Find max
    int stride = 2;
    int shift = 1;
    int slice = len;

    while (1)
    {
        // Find slice
        int tmp = slice / 2;
        if (slice % 2)
        {
            tmp = tmp + 1;
        }

        slice = tmp;

        // Run kernel
        int blocks = slice / CNN_THREAD_PER_BLOCK;
        if (slice % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }
        cnn_max_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(buf, len, slice,
                                                         stride, shift);

        if (slice == 1)
        {
            break;
        }

        // Find new stride, shift
        stride *= 2;
        shift *= 2;
    }

    // Copy result
    hipMemcpy(maxPtr, buf, sizeof(float), hipMemcpyDeviceToHost);
}

__device__ float add_cu(float src1, float src2) { return src1 + src2; }

__global__ void cnn_sum_kernel(float* vec, int len, int slice, int stride,
                               int shift)
{
    int sliceIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (sliceIndex >= slice)
    {
        return;
    }

    int head = sliceIndex * stride;
    int cmp = head + shift;
    if (cmp < len)
    {
        vec[head] = add_cu(vec[head], vec[cmp]);
    }
}

void cnn_sum_gpu(float* sumPtr, float* vec, int len, float* buf)
{
    // Copy memory
    hipMemcpy(buf, vec, len * sizeof(float), hipMemcpyDeviceToDevice);

    // Find sum
    int stride = 2;
    int shift = 1;
    int slice = len;

    while (1)
    {
        // Find slice
        int tmp = slice / 2;
        if (slice % 2)
        {
            tmp = tmp + 1;
        }

        slice = tmp;

        // Run kernel
        int blocks = slice / CNN_THREAD_PER_BLOCK;
        if (slice % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }
        cnn_sum_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(buf, len, slice,
                                                         stride, shift);

        if (slice == 1)
        {
            break;
        }

        // Find new stride, shift
        stride *= 2;
        shift *= 2;
    }

    // Copy result
    hipMemcpy(sumPtr, buf, sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void cnn_add_kernel(float* dst, float* src, int len, float addend)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len)
    {
        return;
    }

    dst[index] = src[index] + addend;
}

void cnn_add_gpu(float* dst, float* src, int len, float addend)
{
    int blocks = len / CNN_THREAD_PER_BLOCK;
    if (len % CNN_THREAD_PER_BLOCK)
    {
        blocks += 1;
    }

    cnn_add_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len, addend);
}

__global__ void cnn_exp_kernel(float* dst, float* src, int len)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len)
    {
        return;
    }

    dst[index] = __expf(src[index]);
}

void cnn_exp_gpu(float* dst, float* src, int len)
{
    int blocks = len / CNN_THREAD_PER_BLOCK;
    if (len % CNN_THREAD_PER_BLOCK)
    {
        blocks += 1;
    }

    cnn_exp_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len);
}

__global__ void cnn_div_kernel(float* dst, float* src, int len, float divider)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len)
    {
        return;
    }

    dst[index] = src[index] / divider;
}

void cnn_div_gpu(float* dst, float* src, int len, float divider)
{
    int blocks = len / CNN_THREAD_PER_BLOCK;
    if (len % CNN_THREAD_PER_BLOCK)
    {
        blocks += 1;
    }

    cnn_div_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len, divider);
}

__global__ void cnn_fmaxf_kernel(float* dst, float* src, int len, float num)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len)
    {
        return;
    }

    dst[index] = fmaxf(src[index], num);
}

void cnn_fmaxf_gpu(float* dst, float* src, int len, float num)
{
    int blocks = len / CNN_THREAD_PER_BLOCK;
    if (len % CNN_THREAD_PER_BLOCK)
    {
        blocks += 1;
    }

    cnn_fmaxf_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len, num);
}

__global__ void cnn_smax_grad_kernel(float* dst, float* cache, int len)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= len || j >= len)
    {
        return;
    }

    dst[i * len + j] = cache[i] * ((float)(i == j) - cache[j]);
}

void cnn_smax_grad_gpu(float* dst, float* cache, int len)
{
    int blocks = len / CNN_THREAD_PER_BLOCK_2D;
    if (len % CNN_THREAD_PER_BLOCK_2D)
    {
        blocks += 1;
    }

    dim3 blk(CNN_THREAD_PER_BLOCK_2D, CNN_THREAD_PER_BLOCK_2D);
    dim3 grid(blocks, blocks);

    cnn_smax_grad_kernel<<<grid, blk>>>(dst, cache, len);
}

__global__ void cnn_relu_grad_kernel(float* dst, float* src, int len)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len)
    {
        return;
    }

    dst[index] = (src[index] < 0.0f) ? 0 : 1;
}

void cnn_relu_grad_gpu(float* dst, float* src, int len)
{
    int blocks = len / CNN_THREAD_PER_BLOCK;
    if (len % CNN_THREAD_PER_BLOCK)
    {
        blocks += 1;
    }

    cnn_relu_grad_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len);
}
