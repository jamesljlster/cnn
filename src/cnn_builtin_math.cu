#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "cnn_builtin_math_cu.h"
#include "cnn_cudef.h"

#define CNN_SCALAR_ACTIV_IMPL(name, fwProc, bpProc)                           \
    __global__ void cnn_##name##_kernel(float* dst, float* src, int len)      \
    {                                                                         \
        int index = blockIdx.x * blockDim.x + threadIdx.x;                    \
        if (index >= len)                                                     \
        {                                                                     \
            return;                                                           \
        }                                                                     \
                                                                              \
        fwProc                                                                \
    }                                                                         \
                                                                              \
    void cnn_##name##_gpu(float* dst, float* src, int len)                    \
    {                                                                         \
        int blocks = len / CNN_THREAD_PER_BLOCK;                              \
        if (len % CNN_THREAD_PER_BLOCK)                                       \
        {                                                                     \
            blocks += 1;                                                      \
        }                                                                     \
                                                                              \
        cnn_##name##_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len); \
    }                                                                         \
                                                                              \
    __global__ void cnn_##name##_grad_kernel(                                 \
        float* gradOut, float* gradIn, float* src, int len, float* cache)     \
    {                                                                         \
        int index = blockIdx.x * blockDim.x + threadIdx.x;                    \
        if (index >= len)                                                     \
        {                                                                     \
            return;                                                           \
        }                                                                     \
                                                                              \
        bpProc                                                                \
    }                                                                         \
                                                                              \
    void cnn_##name##_grad_gpu(float* gradOut, float* gradIn, float* src,     \
                               int len, float* cache)                         \
    {                                                                         \
        int blocks = len / CNN_THREAD_PER_BLOCK;                              \
        if (len % CNN_THREAD_PER_BLOCK)                                       \
        {                                                                     \
            blocks += 1;                                                      \
        }                                                                     \
                                                                              \
        cnn_##name##_grad_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(           \
            gradOut, gradIn, src, len, cache);                                \
    }

#ifdef __cplusplus
extern "C"
{
#endif

    __device__ float max_cu(float src1, float src2)
    {
        return (src1 > src2) ? src1 : src2;
    }

    __global__ void cnn_max_kernel(float* vec, int len, int slice, int stride,
                                   int shift)
    {
        int sliceIndex = blockIdx.x * blockDim.x + threadIdx.x;
        if (sliceIndex >= slice)
        {
            return;
        }

        int head = sliceIndex * stride;
        int cmp = head + shift;
        if (cmp < len)
        {
            vec[head] = max_cu(vec[head], vec[cmp]);
        }
    }

    void cnn_max_gpu(float* maxPtr, float* vec, int len, float* buf)
    {
        // Copy memory
        hipMemcpy(buf, vec, len * sizeof(float), hipMemcpyDeviceToDevice);

        // Find max
        int stride = 2;
        int shift = 1;
        int slice = len;

        while (1)
        {
            // Find slice
            int tmp = slice / 2;
            if (slice % 2)
            {
                tmp = tmp + 1;
            }

            slice = tmp;

            // Run kernel
            int blocks = slice / CNN_THREAD_PER_BLOCK;
            if (slice % CNN_THREAD_PER_BLOCK)
            {
                blocks += 1;
            }
            cnn_max_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(buf, len, slice,
                                                             stride, shift);

            if (slice == 1)
            {
                break;
            }

            // Find new stride, shift
            stride *= 2;
            shift *= 2;
        }

        // Copy result
        hipMemcpy(maxPtr, buf, sizeof(float), hipMemcpyDeviceToHost);
    }

    __device__ float add_cu(float src1, float src2) { return src1 + src2; }

    __global__ void cnn_sum_kernel(float* vec, int len, int slice, int stride,
                                   int shift)
    {
        int sliceIndex = blockIdx.x * blockDim.x + threadIdx.x;
        if (sliceIndex >= slice)
        {
            return;
        }

        int head = sliceIndex * stride;
        int cmp = head + shift;
        if (cmp < len)
        {
            vec[head] = add_cu(vec[head], vec[cmp]);
        }
    }

    void cnn_sum_gpu(float* sumPtr, float* vec, int len, float* buf)
    {
        // Copy memory
        hipMemcpy(buf, vec, len * sizeof(float), hipMemcpyDeviceToDevice);

        // Find sum
        int stride = 2;
        int shift = 1;
        int slice = len;

        while (1)
        {
            // Find slice
            int tmp = slice / 2;
            if (slice % 2)
            {
                tmp = tmp + 1;
            }

            slice = tmp;

            // Run kernel
            int blocks = slice / CNN_THREAD_PER_BLOCK;
            if (slice % CNN_THREAD_PER_BLOCK)
            {
                blocks += 1;
            }
            cnn_sum_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(buf, len, slice,
                                                             stride, shift);

            if (slice == 1)
            {
                break;
            }

            // Find new stride, shift
            stride *= 2;
            shift *= 2;
        }

        // Copy result
        hipMemcpy(sumPtr, buf, sizeof(float), hipMemcpyDeviceToHost);
    }

    __global__ void cnn_add_kernel(float* dst, float* src, int len,
                                   float addend)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        dst[index] = src[index] + addend;
    }

    void cnn_add_gpu(float* dst, float* src, int len, float addend)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_add_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len, addend);
    }

    __global__ void cnn_exp_kernel(float* dst, float* src, int len)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        dst[index] = expf(src[index]);
    }

    void cnn_exp_gpu(float* dst, float* src, int len)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_exp_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len);
    }

    __global__ void cnn_mul_kernel(float* dst, float* src, int len,
                                   float multipiler)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        dst[index] = src[index] * multipiler;
    }

    void cnn_mul_gpu(float* dst, float* src, int len, float multipiler)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_mul_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len,
                                                         multipiler);
    }

    __global__ void cnn_div_kernel(float* dst, float* src, int len,
                                   float divider)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        dst[index] = src[index] / divider;
    }

    void cnn_div_gpu(float* dst, float* src, int len, float divider)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_div_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len,
                                                         divider);
    }

    __global__ void cnn_fminf_kernel(float* dst, float* src, int len, float num)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        dst[index] = fminf(src[index], num);
    }

    void cnn_fminf_gpu(float* dst, float* src, int len, float num)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_fminf_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len, num);
    }

    __global__ void cnn_fmaxf_kernel(float* dst, float* src, int len, float num)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        dst[index] = fmaxf(src[index], num);
    }

    void cnn_fmaxf_gpu(float* dst, float* src, int len, float num)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_fmaxf_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src, len, num);
    }

    __global__ void cnn_elemwise_add_kernel(float* dst, float* src1,
                                            float* src2, int len)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        dst[index] = src1[index] + src2[index];
    }

    void cnn_elemwise_add_gpu(float* dst, float* src1, float* src2, int len)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_elemwise_add_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(dst, src1,
                                                                  src2, len);
    }

    __global__ void cnn_elemwise_product_kernel(float* dst, float* src1,
                                                float* src2, int len)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= len)
        {
            return;
        }

        dst[index] = src1[index] * src2[index];
    }

    void cnn_elemwise_product_gpu(float* dst, float* src1, float* src2, int len)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK;
        if (len % CNN_THREAD_PER_BLOCK)
        {
            blocks += 1;
        }

        cnn_elemwise_product_kernel<<<blocks, CNN_THREAD_PER_BLOCK>>>(
            dst, src1, src2, len);
    }

    __global__ void cnn_smax_grad_kernel(float* dst, float* cache, int len)
    {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= len || j >= len)
        {
            return;
        }

        dst[i * len + j] = cache[i] * ((float)(i == j) - cache[j]);
    }

    void cnn_smax_grad_gpu(float* dst, float* cache, int len)
    {
        int blocks = len / CNN_THREAD_PER_BLOCK_2D;
        if (len % CNN_THREAD_PER_BLOCK_2D)
        {
            blocks += 1;
        }

        dim3 blk(CNN_THREAD_PER_BLOCK_2D, CNN_THREAD_PER_BLOCK_2D);
        dim3 grid(blocks, blocks);

        cnn_smax_grad_kernel<<<grid, blk>>>(dst, cache, len);
    }

    CNN_SCALAR_ACTIV_IMPL(                                         //
        relu,                                                      //
        dst[index] = fmaxf(src[index], 0.0f);                      //
        ,                                                          //
        gradOut[index] = (src[index] < 0.0f) ? 0 : gradIn[index];  //
    )

    CNN_SCALAR_ACTIV_IMPL(                                     //
        swish,                                                 //
        dst[index] = src[index] / (1.0f + expf(-src[index]));  //
        ,                                                      //
        if (src[index] == 0.0f)                                //
        {                                                      //
            gradOut[index] = 0.5;                              //
        }                                                      //
        else                                                   //
        {                                                      //
            gradOut[index] = (cache[index] + (cache[index] / src[index]) *
                                                 (1.0f - cache[index])) *
                             gradIn[index];  //
        }                                    //
    )

    CNN_SCALAR_ACTIV_IMPL(                               //
        sigmoid,                                         //
        dst[index] = 1.0f / (1.0f + expf(-src[index]));  //
        ,                                                //
        gradOut[index] = cache[index] * (1.0 - cache[index]) *
                         gradIn[index];  //
    )

    CNN_SCALAR_ACTIV_IMPL(                                        //
        tanh,                                                     //
        dst[index] = 2.0 / (1.0 + exp(-2.0 * src[index])) - 1.0;  //
        ,                                                         //
        gradOut[index] = (1.0 - cache[index] * cache[index]) *
                         gradIn[index];  //
    )

    CNN_SCALAR_ACTIV_IMPL(                                            //
        gaussian,                                                     //
        dst[index] = exp(-src[index] * src[index] * 0.5);             //
        ,                                                             //
        gradOut[index] = -src[index] * cache[index] * gradIn[index];  //
    )

    CNN_SCALAR_ACTIV_IMPL(  //
        bent_identity,      //
        dst[index] = (sqrt(src[index] * src[index] + 1.0) - 1.0) / 2.0 +
                     src[index];  //
        ,                         //
        gradOut[index] =
            (src[index] / (2.0 * sqrt(src[index] * src[index] + 1.0)) + 1.0) *
            gradIn[index];  //
    )

    CNN_SCALAR_ACTIV_IMPL(                                          //
        softplus,                                                   //
        dst[index] = log1p(exp(src[index]));                        //
        ,                                                           //
        gradOut[index] = gradIn[index] / (1.0 + exp(-src[index]));  //
    )

    CNN_SCALAR_ACTIV_IMPL(                                   //
        softsign,                                            //
        dst[index] = src[index] / (1.0 + fabs(src[index]));  //
        ,                                                    //
        float tmp = 1.0 + fabs(src[index]);                  //
        gradOut[index] = gradIn[index] / (tmp * tmp);        //
    )

    CNN_SCALAR_ACTIV_IMPL(                              //
        sinc,                                           //
        if (src[index] == 0.0)                          //
        {                                               //
            dst[index] = 1.0;                           //
        }                                               //
        else                                            //
        {                                               //
            dst[index] = sin(src[index]) / src[index];  //
        }                                               //
        ,                                               //
        if (src[index] == 0.0)                          //
        {                                               //
            gradOut[index] = 0.0;                       //
        }                                               //
        else                                            //
        {                                               //
            gradOut[index] = ((cos(src[index]) / src[index]) -
                              (sin(src[index]) / (src[index] * src[index]))) *
                             gradIn[index];  //
        }                                    //
    )

    CNN_SCALAR_ACTIV_IMPL(                                 //
        sin,                                               //
        dst[index] = sin(src[index]);                      //
        ,                                                  //
        gradOut[index] = cos(src[index]) * gradIn[index];  //
    )

    CNN_SCALAR_ACTIV_IMPL(               //
        identity,                        //
        dst[index] = src[index];         //
        ,                                //
        gradOut[index] = gradIn[index];  //
    )

#ifdef __cplusplus
}
#endif
